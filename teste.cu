
#include <hip/hip_runtime.h>
#include <iostream>
#include <string>		// Necessário para usar strings
using namespace std;

int main() {
  string text = "c0f21f95d6f5ba2651c599ad9fc847c0923687f6006635e46313335011e152ce1eac3396ba03d15a625d9649a4572cc8dbebcd6810bea0351846bb262a78ab5b";

  int textSize = text.length();
  
  /*string hex = "";
  hex += text[0] ;
  hex += text[1];
  hex += text[2];
  hex += text[3];

  std::cout << hex << "\n";*/

  //FFFF
  string newText;
  int i = 0;
  while(i < textSize / 4){
    string hex = "";
    hex += text[0] ;
    hex += text[i + 1];
    hex += text[i + 2];
    hex += text[i + 3];
    
    //std::cout << hex << "\n";

    uint16_t num = std::stoi( hex, nullptr, 16 );
    uint8_t array[sizeof(num)];

    newText[i] = num;
    
    i = i + 4;
  }

  std::cout << "newText " << newText << "\n";


}